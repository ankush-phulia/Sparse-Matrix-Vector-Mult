#include "hip/hip_runtime.h"
#include "Util.h"

#define THREADS_PER_BLOCK 1024

int N;
int comm_size, proc_Id;
int dim, num_rows; //dim of data, number of rows the proc has

vector<int> indices, ptrs;
vector<int> data; //sparse matrix ka stuff
vector<int> vec;  //vector
long long int *own_output, *output;
vector<int> own_mat_indices, own_mat_ptrs;
vector<int> own_mat_data; //own_matrix stuff

string out_file = "Output_";

__device__ long long int multRow(int noOfElems, int *colIndices, int *nonZeroElems, int *vecTOR)
{
    long long int sum = 0;
    for (int j = 0; j < noOfElems; j++)
    {
		long long int num1 = nonZeroElems[j];
		long long int num2 = vecTOR[colIndices[j]];
		sum += (num1) * (num2);
    }
    return sum;
}

__global__ void multKernel(int *firstElemsRows, int *colIndices, int *nonZeroElems, int numRows, int *vecTOR, long long int *output)
{
    int currRow = blockIdx.x * blockDim.x + threadIdx.x;
    if (currRow < numRows)
    {
		int rowStart = firstElemsRows[currRow];
		int rowEnd = firstElemsRows[currRow + 1];
		output[currRow] = multRow(rowEnd - rowStart, colIndices + rowStart, nonZeroElems + rowStart, vecTOR);
    }
}

void getInput(char *in_file)
{
    if (proc_Id == 0)
    {
		//get input in proc 0
		ifstream f_in;
		f_in.open(in_file);
		//headers, dim and stuff
		string junk, not_junk;
		string temp;
		int data_item;
		int x, y;
		f_in >> junk >> not_junk;
		f_in >> junk >> dim >> not_junk;
		f_in >> temp;

		int xold = -1;
		while (temp[0] != 'B')
		{
			x = atoi(temp.c_str());
			f_in >> y >> data_item;
			data.push_back(data_item);
			indices.push_back(y);
			if (x != xold)
			{
			int diff = x - xold - 1;
			while (diff--)
			{
				ptrs.push_back(indices.size() - 1);
			}

			ptrs.push_back(indices.size() - 1);
			xold = x;
			}
			f_in >> temp;
		}
		int endIndex = ptrs.size();
		int differ = dim - endIndex;
		while (differ--)
			ptrs.push_back(indices.size());

		vec.resize(dim);
		for (int i = 0; i < dim; i++)
		{
			f_in >> data_item;
			vec[i] = data_item;
		}

		f_in.close();

		//pick up left over rows for proc 0
		num_rows += dim % comm_size;
    }

    //tell everyone about their load
    MPI_Bcast(&dim, 1, MPI_INT, 0, MPI_COMM_WORLD);

    //send the vector
    vec.resize(dim);
    MPI_Bcast(&vec[0], dim, MPI_INT, 0, MPI_COMM_WORLD);

    int chunk = dim / comm_size;

    num_rows += chunk;
    //prepare the output vector
    //send the matrix rows

    if (proc_Id != 0)
    {
		own_mat_ptrs.resize(num_rows + 1);
		MPI_Recv(&own_mat_ptrs[0], own_mat_ptrs.size(), MPI_INT, 0, proc_Id, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		own_mat_data.resize(own_mat_ptrs.back());
		own_mat_indices.resize(own_mat_ptrs.back());
		MPI_Recv(&own_mat_indices[0], own_mat_indices.size(), MPI_INT, 0, proc_Id, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		MPI_Recv(&own_mat_data[0], own_mat_data.size(), MPI_INT, 0, proc_Id, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
    else
    {
		int next_index = num_rows;
		int next = ptrs[next_index];
		int temp, old_index;
		vector<int> temp_ptr, temp_indices;
		vector<int> temp_data;
		//own stuff
		own_mat_indices = sub(indices, 0, next);
		own_mat_data = sub(data, 0, next);
		own_mat_ptrs = sub(ptrs, 0, num_rows);
		own_mat_ptrs.push_back(own_mat_indices.size());
		//send load to others
		for (int i = 1; i < comm_size - 1; i++)
		{
			old_index = next_index;
			next_index += chunk;
			temp = next;
			next = ptrs[next_index];
			temp_data = sub(data, temp, next);
			temp_indices = sub(indices, temp, next);
			temp_ptr = sub(ptrs, old_index, next_index);
			mapped_subtract(temp_ptr, temp);
			temp_ptr.push_back(temp_indices.size());
			MPI_Send(&temp_ptr[0], temp_ptr.size(), MPI_INT, i, i, MPI_COMM_WORLD);
			MPI_Send(&temp_indices[0], temp_indices.size(), MPI_INT, i, i, MPI_COMM_WORLD);
			MPI_Send(&temp_data[0], temp_data.size(), MPI_INT, i, i, MPI_COMM_WORLD);
		}
		//final process' load
		temp_data = sub(data, next, data.size());
		temp_indices = sub(indices, next, indices.size());
		temp_ptr = sub(ptrs, next_index, ptrs.size());
		mapped_subtract(temp_ptr, next);
		temp_ptr.push_back(temp_indices.size());
		MPI_Send(&temp_ptr[0], temp_ptr.size(), MPI_INT, comm_size - 1, comm_size - 1, MPI_COMM_WORLD);
		// cout << "sending to last2 "<< temp_indices.size() << endl;
		MPI_Send(&temp_indices[0], temp_indices.size(), MPI_INT, comm_size - 1, comm_size - 1, MPI_COMM_WORLD);
		// cout << "sending to last3 "<< temp_data.size() << endl;
		MPI_Send(&temp_data[0], temp_data.size(), MPI_INT, comm_size - 1, comm_size - 1, MPI_COMM_WORLD);
    }
}

void getOutput()
{
    if (proc_Id == 0)
    {
		output = new long long int[dim];
		memcpy(output, own_output, num_rows * sizeof(long long int));
		int len, totallen = num_rows;
		for (int i = 1; i < comm_size; i++)
		{
			MPI_Recv(&len, 1, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			MPI_Recv(output + totallen, len, MPI_LONG_LONG_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			totallen += len;
		}
    }
    else
    {
		for (int i = 1; i < comm_size; i++)
		{
			MPI_Send(&num_rows, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
			MPI_Send(own_output, num_rows, MPI_LONG_LONG_INT, 0, 0, MPI_COMM_WORLD);
		}
    }
}

void wrapperForCuda()
{
    int dimension = dim;
    //Get the number of rows being handled by the current process
    int numRows = num_rows;

    //Each partition's rows[i], colsIndices[i] and values[i]
    int *currPartitionFirstElemsRows;
    currPartitionFirstElemsRows = &own_mat_ptrs[0];

    int *currPartitionColIndices;
    currPartitionColIndices = &own_mat_indices[0];

    int *currPartitionNonZeroElems;
    currPartitionNonZeroElems = &own_mat_data[0];

    //Commom vector for all processes
    int *vecTOR;
    vecTOR = &vec[0];

    //Device copies for computation
    int *devCurrPartitionFirstElemsRows;
    int *devCurrPartitionColIndices;
    int *devCurrPartitionNonZeroElems;
    int *devVec;
    long long int *devFinalVec;

    int size1 = own_mat_ptrs.size() * sizeof(int);
    int size2 = own_mat_indices.size() * sizeof(int);
    int size3 = own_mat_data.size() * sizeof(int);

    //Current process's computed output
    own_output = (long long int *)malloc(sizeof(long long int) * numRows);

    //once
    hipMalloc((void **)&devFinalVec, numRows * sizeof(long long int));
    hipMalloc((void **)&devVec, dimension * sizeof(int));

    N = numRows;

    hipMalloc((void **)&devCurrPartitionFirstElemsRows, size1);
    hipMalloc((void **)&devCurrPartitionColIndices, size2);
    hipMalloc((void **)&devCurrPartitionNonZeroElems, size3);

    hipMemcpy(devCurrPartitionFirstElemsRows, currPartitionFirstElemsRows, size1, hipMemcpyHostToDevice);
    hipMemcpy(devCurrPartitionColIndices, currPartitionColIndices, size2, hipMemcpyHostToDevice);
    hipMemcpy(devCurrPartitionNonZeroElems, currPartitionNonZeroElems, size3, hipMemcpyHostToDevice);
    hipMemcpy(devVec, vecTOR, dimension * sizeof(int), hipMemcpyHostToDevice);

    //Tuning for the problem size
    int blocks;
    int thrds;
    if (num_rows < THREADS_PER_BLOCK)
    {
	blocks = 1;
	thrds = num_rows;
    }
    else
    {
	thrds = THREADS_PER_BLOCK;
	blocks = (num_rows / thrds) + 1;
    }

    multKernel<<<blocks, thrds>>>(devCurrPartitionFirstElemsRows, devCurrPartitionColIndices, devCurrPartitionNonZeroElems, numRows, devVec, devFinalVec);

    hipMemcpy(own_output, devFinalVec, numRows * sizeof(long long int), hipMemcpyDeviceToHost);
}

void computeForEachProcess()
{
    wrapperForCuda();
}

void fileWrite(char *name)
{
    if (proc_Id == 0)
    {
	ofstream f_out;
	f_out.open(name);
	for (int i = 0; i < dim; i++)
	{
	    f_out << output[i] << '\n';
	}
	f_out.close();
    }
}
int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &proc_Id);

    num_rows = 0;
    string in_file = argv[1];
    getInput(argv[1]);
	//--- Till this point every process has a copy of the vector and a CSR representation of its slice of the matrix
    computeForEachProcess();
    
	//Gathering outputs from each process
	getOutput();

    fileWrite(argv[2]);
    MPI_Finalize();
}